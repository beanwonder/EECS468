/* Matrix multiplication: C = A * B.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
// includes, project
#include <cutil.h>

// includes, kernels
#include <matrixmul_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P);
void MatrixInversionOnDevice(Matrix Mtemp_h , int size , Matrix Mtemp1_h);
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	Matrix  M;
	Matrix  N;
	Matrix  P;
	int errorM = 0, errorN = 0;
	
	srand(52);
	
	if(argc != 5 && argc != 4) 
	{
		// Allocate and initialize the matrices
		M  = AllocateMatrix(rand() % 1024, rand() % 1024, 1);
		N  = AllocateMatrix(M.width, rand() % 1024, 1);
		P  = AllocateMatrix(M.height, N.width, 0);
	}
	else
	{
		// Allocate and read in matrices from disk
		int* params = NULL; //(int*)malloc(3 * sizeof(int));
		unsigned int data_read = 3;
		cutReadFilei(argv[1], &params, &data_read, true);
		if(data_read != 3){
			printf("Error reading parameter file\n");
			return 1;
		}

		M  = AllocateMatrix(params[0], params[1], 0);
		N  = AllocateMatrix(params[1], params[2], 0);		
		P  = AllocateMatrix(params[0], params[2], 0);
		errorM = ReadFile(&M, argv[2]);
		errorN = ReadFile(&N, argv[3]);
		if(errorM  || errorN )
		{
			printf("Error reading input files %d, %d\n", errorM, errorN);
			return 1;
		}
	}

	// M * N on the device
	// MatrixMulOnDevice(M, N, P);
    Matrix res = AllocateMatrix(M.height, M.width, 0);
    MatrixInversionOnDevice(M, M.height, res);
    
	printf("GPU computation complete\n");
	// compute the matrix multiplication on the CPU for comparison
	// Matrix reference = AllocateMatrix(P.height, P.width, 0);
	// computeGold(reference.elements, M.elements, N.elements, M.height, M.width, N.width);
        
	// printf("CPU computation complete\n");
	// in this case check if the result is equivalent to the expected soluion
	// CUTBoolean res = cutComparefe(reference.elements, P.elements, P.height*P.width, 0.001f);
	// printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

    /* 
   	if(argc == 5)
	{
		WriteFile(P, argv[4]);
	}
	else if(argc == 2)
	{
		WriteFile(P, argv[1]);
	} 
    */
    /*
    // output M
    printf("original M :\n");
    for (int i=0; i < M.height; ++i) {
        for (int j=0; j < M.width; ++j) {
            std::cout << M.elements[i * M.height + j] << ' ';
        }
        std::cout << '\n';
    }

    std::cout << ("inversion M :\n");
    for (int i=0; i < res.height; ++i) {
        for (int j=0; j < res.width; ++j) {
            std::cout << res.elements[i * res.height + j] << ' ';
        }
        std::cout << '\n';
    }
    */
	// Free matrices
	FreeMatrix(&M);
	FreeMatrix(&N);
	FreeMatrix(&P);
	return 0;
}

void display_matrix(Matrix M) {
    std::cout << ("Matrix display :\n");
    for (int i=0; i < M.height; ++i) {
        for (int j=0; j < M.width; ++j) {
            std::cout << M.elements[i * M.width + j] << ' ';
        }
        std::cout << '\n';
    }
}

// MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
// this function run on cpu and calculate matrixInversion
void MatrixInversionOnDevice(Matrix Mtemp_h , int size , Matrix Mtemp1_h) 
{ 
    
    //Memory allocation on the device 
    display_matrix(Mtemp_h);
    Matrix MM_host = AllocateMatrix(Mtemp_h.height, Mtemp_h.width*2, 0);
    Matrix MM_device = AllocateDeviceMatrix(MM_host);
    
    for (int i=0; i < size; ++i) {
        for (int j=0; j < size; ++j) {
            MM_host.elements[i * MM_host.width + j] = Mtemp_h.elements[i * Mtemp_h.width + j];
        }
    }
    
    for (int i=0; i < size; ++i) {
        MM_host.elements[i * MM_host.width + size + i] = 1;
    }
    
    display_matrix(MM_host);

    CopyToDeviceMatrix(MM_device, MM_host);
    
    //Kernel call 
    // MatrixInversionKernel1<<<dimGrid, dimBlock>>>(Ma, Mb, numvar); 
    
    // MM_device augmented matrix
    dim3 dimGrid1(1);
    dim3 dimGridn(size);
    dim3 dimBlockn(size);
    for (int j=0; j < size; ++j) {
        // addup
        addupKernel<<<dimGrid1, dimBlockn>>>(MM_device, size, j);
        // step2
        fixRowKernel<<<dimGrid1, dimBlockn>>>(MM_device, size, j);
        // step3
        fixColumnKernel<<<dimGridn, dimBlockn>>>(MM_device, size, j);
    }
    
    // Coping data to host from device 
    CopyFromDeviceMatrix(MM_host, MM_device);
    //Deallocating memory on the device 
    display_matrix(MM_host);
    for (int i=0; i < size; ++i) {
        for (int j=0; j < size; ++j) {
            Mtemp1_h.elements[i * Mtemp1_h.width + j] = MM_host.elements[i * MM_host.width + size + j];
        }
    }
    FreeDeviceMatrix(&MM_device); 
    //FreeDeviceMatrix(&Mb); 
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P)
{
	// Load M and N to the device
	Matrix Md = AllocateDeviceMatrix(M);
	CopyToDeviceMatrix(Md, M);
	Matrix Nd = AllocateDeviceMatrix(N);
	CopyToDeviceMatrix(Nd, N);

	// Allocate P on the device
	Matrix Pd = AllocateDeviceMatrix(P);
	CopyToDeviceMatrix(Pd, P); // Clear memory

	// Setup the execution configuration
    const int TILE_DIM = 32;
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid((Pd.width/TILE_DIM)+1, (Pd.height/TILE_DIM)+1);

    // Launch the device computation threads!
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    // Read P from the device
    CopyFromDeviceMatrix(P, Pd); 

	// Free device matrices
	FreeDeviceMatrix(&Md);
	FreeDeviceMatrix(&Nd);
	FreeDeviceMatrix(&Pd);
}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a device matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
//  If init == 2, initialize matrix parameters, but do not allocate memory 
Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand()*3 / (float)RAND_MAX);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

// Read a floating point matrix in from file
// Returns zero if the number of elements read is 
//  equals M.height * M.width, and 1 otherwise
int ReadFile(Matrix* M, char* file_name)
{
	unsigned int data_read = M->height*M->width;
	cutReadFilef(file_name, &(M->elements), &data_read, true);
	return (data_read != (M->height * M->width));
}

// Write a 16x16 floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
    cutWriteFilef(file_name, M.elements, M.width*M.height,
                       0.0001f);
}
